#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <fcntl.h>
#include <unistd.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define EVENT_LENGTH 4
#include <string.h>
// #define NUM_OF_DATA_POINTS 2000000000
// #define BLOCKS (NUM_OF_DATA_POINTS/2)*2
const uint64_t NUM_OF_DATA_POINTS =550000000;
 void randomStream(int N,void * stream) {

//   Define N times char array
 char *copyStream = (char *) malloc(sizeof(char) * (N));
    printf("Address of copyStream: %p\n", copyStream);
  for (uint64_t i = 0; i < N; i++)
  {
    //   printf("%d\n",i);
    copyStream[i] = rand ();
    // printf("%d\n", copyStream[i]);
  }
  printf("Memory allocated\n");
    memcpy(stream, copyStream, N);
    free(copyStream);
}

__global__ void square(float * data){
    //  Blocks: 10000,20000
    // 0,0
    // 0,1
    // 0,2
    uint64_t idx = (threadIdx.x + (blockIdx.x * blockDim.x))*5;
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < NUM_OF_DATA_POINTS*2)
    {
        for (int index=idx;index<idx+5;index++)
        {
        float x = data[idx] * data[idx];
        // printf(" Data: %f Squared: %f ID:%d, Thread :%d , Block: %d , Block Dim: %d \n",data[idx],x, idx,threadIdx.x,blockIdx.x,blockDim.x);
        data[idx] = x;
        }
        

    }
}

int main(){
    // Amount of data we want to get from urandom
    printf("Setting up the array of random numbers\n");
    uint16_t *random_data=(uint16_t *) malloc(NUM_OF_DATA_POINTS*2*sizeof(uint16_t));
    // if (!random_data) { perror("malloc arr"); exit(EXIT_FAILURE); };
    printf("Address of random_data: %p\n", random_data);
    printf("Getting data from urandom...\n");
    uint64_t bytesToBeRead = sizeof(uint16_t)*2*NUM_OF_DATA_POINTS;
    printf("Bytes to be read: %llu\n", bytesToBeRead);
    // Read data from urandom
   // int fd = open("/dev/urandom", O_RDONLY);
    //if (fd < 0) { perror("open"); exit(EXIT_FAILURE); };
    //ssize_t bytesRead = read(fd, random_data, bytesToBeRead);
    //if (bytesRead < 0) { perror("read"); exit(EXIT_FAILURE); };
    randomStream(bytesToBeRead,random_data);
    // printf("Bytes read: %zu\n", bytesRead);
    // close(fd);
    printf("Closed urandom\n");

    // arc4random_buf(random_data, bytesToBeRead);
    
    // Count
    uint64_t inside_circle = 0;
    uint64_t outside_circle = 0;

    // float *positive_random_data;
    // Convert the data into a float between 0 and 1
    float * positive_random_data=(float *) malloc(NUM_OF_DATA_POINTS*2*sizeof(float));
    // cudaMallocManaged(&positive_random_data, NUM_OF_DATA_POINTS*2*sizeof(float));

    for(uint64_t i = 0; i < NUM_OF_DATA_POINTS*2; i++){
        positive_random_data[i] =(long double)random_data[i] / UINT16_MAX;
    }
    free(random_data);
    printf("Finsihed converting data\n");

    printf("Copying data to GPU\n");
    float *copy;
    hipMalloc((void**)&copy, NUM_OF_DATA_POINTS*2*sizeof(float));
    hipMemcpy(copy,positive_random_data,NUM_OF_DATA_POINTS*2*sizeof(float),hipMemcpyHostToDevice);
    square<<<3906250,1024>>>(copy);
    hipMemcpy(positive_random_data,copy,NUM_OF_DATA_POINTS*2*sizeof(float),hipMemcpyDeviceToHost);
    // cudaDeviceSynchronize();
    printf("Finished squaring y\n");

    // Loop through the data and count the number of points inside the circle
    for (uint64_t i=0; i<NUM_OF_DATA_POINTS; i++) {
        // printf("Indexs: %d, %d\n",i,i+NUM_OF_DATA_POINTS);
        if (positive_random_data[i] + positive_random_data[i+NUM_OF_DATA_POINTS] <= 1) {
            inside_circle++;
        } else {
            outside_circle++;
        }
    }
    printf("Finished adding x\n");




    printf("Inside circle: %llu\n", inside_circle);
    printf("Outside circle: %llu\n", outside_circle);
    printf("Pi is: %f\n", 4.0*(float)(inside_circle)/(float)(inside_circle + outside_circle));



    return 0;
}
