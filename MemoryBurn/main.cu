#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

 

#define N 1e+10
#define MAX_ERR 1e-6


__global__ void multiply(int * a,int b,int n)
{
    int idx = threadIdx.x+ blockIdx.x * blockDim.x;
    if (idx < n)
        a[idx] = a[idx] * b;
    // a[idx] *= b;
}

int main(){
    int *a, *b, *out;
    int *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (int*)calloc( N,sizeof(int));


    // Allocate device memory 
    hipMalloc((void**)&d_a, sizeof(int) * N);
    // int counter=1000;
    while (1){
        // counter--;
        // Transfer data from host to device memory
        hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
        multiply<<<1,N>>>(d_a,2,N);
        hipMemcpy(a, d_a, sizeof(int) * N, hipMemcpyDeviceToHost);
    }

}
